#include <hip/hip_runtime.h>

__global__ void copy_matrix_kernel(const float* A, float* B, int N) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i >= N*N) return;
    B[i] = A[i];
}

// A, B are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const float* A, float* B, int N) {
    int total = N * N;
    int threadsPerBlock = 256;
    int blocksPerGrid = (total + threadsPerBlock - 1) / threadsPerBlock;
    copy_matrix_kernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, N);
    hipDeviceSynchronize();
} 