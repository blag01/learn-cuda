#include <hip/hip_runtime.h>

__global__ void matrix_transpose_kernel(const float* input, float* output, int rows, int cols) {
    int r = threadIdx.y + blockDim.y * blockIdx.y;
    int c = threadIdx.x + blockDim.x * blockIdx.x;

    if (r < rows && c < cols)
        output[c * rows + r] = input[r * cols + c];
}

// input, output are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const float* input, float* output, int rows, int cols) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrix_transpose_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, rows, cols);
    hipDeviceSynchronize();
}