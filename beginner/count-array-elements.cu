#include <hip/hip_runtime.h>

__global__ void count_equal_kernel(const int* input, int* output, int N, int K) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if( i >= N ) return;
    if (input[i] == K) atomicAdd(output, 1);
}

// input, output are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const int* input, int* output, int N, int K) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    
    count_equal_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, N, K);
    hipDeviceSynchronize();
}