#include <hip/hip_runtime.h>

__global__ void invert_kernel(unsigned char* image, int width, int height) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index >= width * height) return;

    for (int i = 0 ; i < 3; i++)
        image[4 * index + i] = 255 - image[4 * index + i];
}
// image_input, image_output are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(unsigned char* image, int width, int height) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (width * height + threadsPerBlock - 1) / threadsPerBlock;

    invert_kernel<<<blocksPerGrid, threadsPerBlock>>>(image, width, height);
    hipDeviceSynchronize();
}